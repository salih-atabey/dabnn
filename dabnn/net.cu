// Copyright 2019 JD.com Inc. JD AI

#include "net.h"

#include <fcntl.h>
#include <glog/logging.h>
#include <sys/mman.h>
#include <chrono>
#include <vector>

#include <common/flatbuffers_helper.h>
#include <common/macros.h>
#include <dabnn/bitpack.h>
#include <dabnn/layers/Add.h>
#include <dabnn/layers/Affine.h>
#include <dabnn/layers/AvePool.h>
#include <dabnn/layers/BinConv.h>
#include <dabnn/layers/Binarize.h>
#include <dabnn/layers/Concat.h>
#include <dabnn/layers/FloatConv.h>
#include <dabnn/layers/MaxPool.h>
#include <dabnn/layers/Relu.h>
#include <dabnn/layers/Shuffle.h>
#include <dabnn/layers/Split.h>
#include <dabnn/layers/PRelu.h>

using std::string;
using std::vector;

namespace bnn {

Net::~Net() {
    for (const auto& pair : mat_map_) {
        auto mat = pair.second;
        if (mat->data_type == DataType::Bit) {
            if (mat->external_memory) {
                hipFree(mat->data);
            }
        } else if (mat->data_type == DataType::Float) {
            if (mat->external_memory) {
                hipFree(mat->data);
            }
        }
    }
}

void Net::read(const std::string &path) {
    auto fd = open(path.c_str(), O_RDONLY);
    if (fd == -1) {
        throw std::invalid_argument("Open file error " + std::to_string(errno));
    }
    size_t fsize = static_cast<size_t>(lseek(fd, 0, SEEK_END));
    auto data = mmap(nullptr, fsize, PROT_READ, MAP_PRIVATE, fd, 0);
    if (data == MAP_FAILED) {
        throw std::invalid_argument("mmap failed, errno = " +
                                    std::to_string(errno));
    }
    read_impl(data);
}

void Net::read_buf(const void *ptr) { read_impl(ptr); }

void Net::read_impl(const void *ptr) {
    const auto model = flatbnn::GetModel(ptr);
    model_ = model;
    prepare();
}

void Net::prepare() {
    BNN_ASSERT(!(strict && !run_fconv), "fconv must be run in strict mode");
    BNN_ASSERT(model_->version() == BNN_LATEST_MODEL_VERSION,
               "The model version should be ", BNN_LATEST_MODEL_VERSION,
               ", got ", model_->version(), " instead.");
    for (const auto &tensor : *model_->inputs()) {
        Shaper::Shape shape(tensor->shape()->begin(), tensor->shape()->end());
        const auto name = tensor->name()->str();

        shaper.AddShape(name, shape);
        add_mat(name, std::make_shared<Mat>(shape[1], shape[2], shape[3],
                                            bnn::DataType::Float));

        input_name_ = name;

        break;
    }

    for (const auto &tensor : *model_->initializers()) {
        if (tensor->data_type() == flatbnn::DataType::Bit) {
            // This shape is the same as that of flatbuffers
            Shaper::Shape shape(tensor->shape()->begin(),
                                tensor->shape()->end());

            const auto name = tensor->name()->str();

            shaper.AddShape(name, shape);

            const auto len = tensor->bin_data()->size();

            // const auto *data = tensor->bin_data()->data();
            uint64_t *data;
            hipMallocManaged((void **)&data, len * sizeof(uint64_t));
            hipMemcpy(data, tensor->bin_data()->data(), len * sizeof(uint64_t), hipMemcpyHostToDevice);
#ifdef __aarch64__
            // TODO: Move it to binconv.cpp
            // 1. More correct
            // 2. Don't need to maintain the the same shape
            if (Shaper::c(shape) % 128 == 0) {
                // Re-arrange the bit order for the optmized bit-packing
                const auto tmp = std::make_shared<Mat>(
                    shape[0], shape[1], shape[2], shape[3],
                    bnn::DataType::Float, false);
                auto *float_data = static_cast<float *>(tmp->data);
                FORZ(i, len) {
                    std::bitset<64> bs(*(data + i));
                    FORZ(j, 64) { float_data[i * 64 + j] = bs[j] ? 1 : -1; }
                }

                add_mat(name, std::make_shared<Mat>(
                                  shape[0], shape[1], shape[2], shape[3],
                                  bnn::DataType::Bit, len, false));
                pack_mat(*tmp, *mat_map_[name]);
            } else {
#endif  // __aarch64__
                add_mat(name, std::make_shared<Mat>(
                                  shape[0], shape[1], shape[2], shape[3],
                                  const_cast<uint64_t *>(data),
                                  bnn::DataType::Bit, len, false));
#ifdef __aarch64__
            }
#endif  // __aarch64__
        } else if (tensor->data_type() == flatbnn::DataType::Float32) {
            Shaper::Shape shape(tensor->shape()->begin(),
                                tensor->shape()->end());
            // const auto *data = tensor->float32_data()->Data();
            float *data;
            
            const auto name = tensor->name()->str();

            shaper.AddShape(name, shape);

            if (shape.size() == 4) {
                // conv weight
                const auto len = shape[0] * shape[1] * shape[2] * shape[3];
                // auto buf = std::make_shared<std::vector<float>>(len);
                // memcpy(buf->data(), data, len * sizeof(float));
                hipMallocManaged((void **)&data, len * sizeof(float));
                hipMemcpy(data, tensor->float32_data()->Data(), len * sizeof(float), hipMemcpyHostToDevice);
                // add_mat(name, std::make_shared<Mat>(
                //                   shape[0], shape[1], shape[2], shape[3],
                //                   const_cast<uint8_t *>(data),
                //                   bnn::DataType::Float, false));
                add_mat(name, std::make_shared<Mat>(
                                  shape[0], shape[1], shape[2], shape[3],
                                  data,
                                  bnn::DataType::Float, false));
            } else if (shape.size() == 1) {
                // bias or affine weight
                // auto buf = std::make_shared<std::vector<float>>(shape[0]);
                // memcpy(buf->data(), data, shape[0] * sizeof(float));
                hipMallocManaged((void **)&data, shape[0] * sizeof(float));
                hipMemcpy(data, tensor->float32_data()->Data(), shape[0] * sizeof(float), hipMemcpyHostToDevice);
                add_mat(name, std::make_shared<Mat>(shape[0], data,
                                                    DataType::Float));
                // float_bufs_.push_back(buf);
            }
        }
    }

    for (const auto *layer : *model_->layers()) {
        VLOG(5) << layer_type_to_str(layer->type());
        const std::string name =
            layer->name() != nullptr ? layer->name()->str() : "";
        switch (layer->type()) {
            case flatbnn::LayerType::FpConv2D: {
                ADD_LAYER(fp_conv2d, Conv, input, strides, dilations, pads,
                          weight, bias, output);
                BNN_ASSERT(pads.size() == 2 ||
                               (pads.size() == 4 && pads[0] == pads[2] &&
                                pads[1] == pads[3]),
                           pads);
                BNN_ASSERT(strides.size() == 2 || (strides.size() == 4 &&
                                                   strides[0] == strides[2] &&
                                                   strides[1] == strides[3]),
                           strides);

                if (run_fconv) {
                    if (bias != "") {
                        layers.push_back(std::make_shared<FloatConv>(
                            get_weak(), name, input, weight, bias, output,
                            pads[0], pads[1], strides[0], strides[1], 1));
                    } else {
                        layers.push_back(std::make_shared<FloatConv>(
                            get_weak(), name, input, weight, output, pads[0],
                            pads[1], strides[0], strides[1], 1));
                    }
                }

                break;
            }
            case flatbnn::LayerType::BinConv2D: {
                ADD_LAYER(bin_conv2d, Conv, input, strides, dilations, pads,
                          weight, output);
                BNN_ASSERT(pads.size() == 2 ||
                               (pads.size() == 4 && pads[0] == pads[2] &&
                                pads[1] == pads[3]),
                           pads);
                BNN_ASSERT(strides.size() == 2 || (strides.size() == 4 &&
                                                   strides[0] == strides[2] &&
                                                   strides[1] == strides[3]),
                           strides);

                layers.push_back(std::make_shared<BinConv>(
                    get_weak(), name, input, weight, output, pads[0], pads[1],
                    strides[0], strides[1]));
                break;
            }
            case flatbnn::LayerType::Affine: {
#ifdef BNN_CHECK_CONSISTENCY
                ADD_LAYER(affine, Affine, input, a, b, output);
                layers.push_back(std::make_shared<Affine>(get_weak(), name,
                                                          input, a, b, output));
#else
                ADD_INPLACE_LAYER(affine, Affine, input, a, b, output);
                layers.push_back(
                    std::make_shared<Affine>(get_weak(), name, input, a, b));
#endif
                break;
            }
            case flatbnn::LayerType::Add: {
#ifdef BNN_CHECK_CONSISTENCY
                ADD_LAYER(add, Eltwise, input1, input2, output)
                layers.push_back(std::make_shared<Add>(get_weak(), name, input1,
                                                       input2, output));
#else
                ADD_INPLACE_LAYER(add, Eltwise, input1, input2, output)
                layers.push_back(
                    std::make_shared<Add>(get_weak(), name, input1, input2));
#endif
                break;
            }
            case flatbnn::LayerType::MaxPool: {
                ADD_LAYER(maxpool, Pool, input, strides, pads, kernel_shape,
                          output);

                layers.push_back(std::make_shared<MaxPool>(
                    get_weak(), name, input, output, kernel_shape[0],
                    kernel_shape[1], pads[0], pads[1], strides[0], strides[1]));
                break;
            }
            case flatbnn::LayerType::AvePool: {
                ADD_LAYER(avepool, Pool, input, strides, pads, kernel_shape,
                          output);

                layers.push_back(std::make_shared<AvePool>(
                    get_weak(), name, input, output, kernel_shape[0],
                    kernel_shape[1], pads[0], pads[1], strides[0], strides[1]));
                break;
            }
            case flatbnn::LayerType::Concat: {
                ADD_LAYER(concat, Concat, inputs, axis, output);
                BNN_ASSERT(axis == 3, "");

                layers.push_back(std::make_shared<Concat>(
                    get_weak(), name, inputs[0], inputs[1], output));
                break;
            }
            case flatbnn::LayerType::Relu: {
                ADD_INPLACE_LAYER(relu, Relu, input, output);

                layers.push_back(
                    std::make_shared<Relu>(get_weak(), name, input));
                break;
            }
            case flatbnn::LayerType::Split: {
                ADD_LAYER_MULTI_OUTPUTS(split, Split, input, outputs);
                layers.push_back(std::make_shared<Split>(
                    get_weak(), name, input, outputs[0], outputs[1]));
                break;
            }
            case flatbnn::LayerType::Shuffle: {
                ADD_INPLACE_LAYER(shuffle, Shuffle, input, output);
                layers.push_back(
                    std::make_shared<Shuffle>(get_weak(), name, input));
                break;
            }
            case flatbnn::LayerType::PRelu: {
                ADD_INPLACE_LAYER(prelu, Eltwise, input, slope, output);
                layers.push_back(
                    std::make_shared<PRelu>(get_weak(), name, input, slope));
                break;
            }
            default: {
                throw std::runtime_error("Not supported op " +
                                         layer_type_to_str(layer->type()));
                break;
            }
        }
    }
}

void Net::run(void *input) {
    BNN_ASSERT(!(strict && !run_fconv), "fconv must be run in strict mode");
    uint64_t t = 0;

    mat_map_[input_name_]->external_memory = true;
    mat_map_[input_name_]->data = input;

    for (const auto &layer : layers) {
        VLOG(5) << layer->to_str();
        layer->forward();
    }

    VLOG(2) << "t = " << t;
    VLOG(2) << "-------";
}

std::shared_ptr<Mat> Net::get_blob(const std::string &name) {
    return mat_map_.at(name);
}

void Net::add_mat(const std::string &name, std::shared_ptr<Mat> mat) {
    mat_map_[name] = mat;
}

std::weak_ptr<Net> Net::get_weak() { return shared_from_this(); }

std::shared_ptr<Net> Net::create() {
    return std::make_shared<make_shared_enabler>();
}

#ifdef BNN_BENCHMARK
void Net::print_time() {
    double total_time = 0;
    for (const auto &kv : layer_time_) {
        total_time += kv.second;
    }
    for (const auto &kv : layer_time_) {
        const auto &name = kv.first;
        const auto &time = kv.second;
        const auto &percent = time / total_time * 100;
        PNT(name, time, percent);
    }
}
#endif

}  // namespace bnn
