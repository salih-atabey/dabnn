// Copyright 2019 JD.com Inc. JD AI

#include "Binarize.h"

#include <dabnn/bitpack.h>
#include <dabnn/net.h>

namespace bnn {

struct pack_64_bitfield_functor
{
    float *fptr_base;
    uint64_t *bptr_base;

    pack_64_bitfield_functor(float* _fptr, uint64_t* _bptr) : fptr_base(_fptr), bptr_base(_bptr) {}

    __host__ __device__
    void operator()(int i)
    {
        float* fptr = fptr_base + i * 64;
        uint64_t* buf = bptr_base + i;
        struct bf {
            unsigned int b0 : 1;
            unsigned int b1 : 1;
            unsigned int b2 : 1;
            unsigned int b3 : 1;
            unsigned int b4 : 1;
            unsigned int b5 : 1;
            unsigned int b6 : 1;
            unsigned int b7 : 1;
            unsigned int b8 : 1;
            unsigned int b9 : 1;
            unsigned int b10 : 1;
            unsigned int b11 : 1;
            unsigned int b12 : 1;
            unsigned int b13 : 1;
            unsigned int b14 : 1;
            unsigned int b15 : 1;
            unsigned int b16 : 1;
            unsigned int b17 : 1;
            unsigned int b18 : 1;
            unsigned int b19 : 1;
            unsigned int b20 : 1;
            unsigned int b21 : 1;
            unsigned int b22 : 1;
            unsigned int b23 : 1;
            unsigned int b24 : 1;
            unsigned int b25 : 1;
            unsigned int b26 : 1;
            unsigned int b27 : 1;
            unsigned int b28 : 1;
            unsigned int b29 : 1;
            unsigned int b30 : 1;
            unsigned int b31 : 1;
            unsigned int b32 : 1;
            unsigned int b33 : 1;
            unsigned int b34 : 1;
            unsigned int b35 : 1;
            unsigned int b36 : 1;
            unsigned int b37 : 1;
            unsigned int b38 : 1;
            unsigned int b39 : 1;
            unsigned int b40 : 1;
            unsigned int b41 : 1;
            unsigned int b42 : 1;
            unsigned int b43 : 1;
            unsigned int b44 : 1;
            unsigned int b45 : 1;
            unsigned int b46 : 1;
            unsigned int b47 : 1;
            unsigned int b48 : 1;
            unsigned int b49 : 1;
            unsigned int b50 : 1;
            unsigned int b51 : 1;
            unsigned int b52 : 1;
            unsigned int b53 : 1;
            unsigned int b54 : 1;
            unsigned int b55 : 1;
            unsigned int b56 : 1;
            unsigned int b57 : 1;
            unsigned int b58 : 1;
            unsigned int b59 : 1;
            unsigned int b60 : 1;
            unsigned int b61 : 1;
            unsigned int b62 : 1;
            unsigned int b63 : 1;
        };

        union bf_u64 {
            bf t;
            uint64_t u64;
        };

        bf_u64 u;
        u.t.b0 = fptr[0] > 0;
        u.t.b1 = fptr[1] > 0;
        u.t.b2 = fptr[2] > 0;
        u.t.b3 = fptr[3] > 0;
        u.t.b4 = fptr[4] > 0;
        u.t.b5 = fptr[5] > 0;
        u.t.b6 = fptr[6] > 0;
        u.t.b7 = fptr[7] > 0;
        u.t.b8 = fptr[8] > 0;
        u.t.b9 = fptr[9] > 0;
        u.t.b10 = fptr[10] > 0;
        u.t.b11 = fptr[11] > 0;
        u.t.b12 = fptr[12] > 0;
        u.t.b13 = fptr[13] > 0;
        u.t.b14 = fptr[14] > 0;
        u.t.b15 = fptr[15] > 0;
        u.t.b16 = fptr[16] > 0;
        u.t.b17 = fptr[17] > 0;
        u.t.b18 = fptr[18] > 0;
        u.t.b19 = fptr[19] > 0;
        u.t.b20 = fptr[20] > 0;
        u.t.b21 = fptr[21] > 0;
        u.t.b22 = fptr[22] > 0;
        u.t.b23 = fptr[23] > 0;
        u.t.b24 = fptr[24] > 0;
        u.t.b25 = fptr[25] > 0;
        u.t.b26 = fptr[26] > 0;
        u.t.b27 = fptr[27] > 0;
        u.t.b28 = fptr[28] > 0;
        u.t.b29 = fptr[29] > 0;
        u.t.b30 = fptr[30] > 0;
        u.t.b31 = fptr[31] > 0;
        u.t.b32 = fptr[32] > 0;
        u.t.b33 = fptr[33] > 0;
        u.t.b34 = fptr[34] > 0;
        u.t.b35 = fptr[35] > 0;
        u.t.b36 = fptr[36] > 0;
        u.t.b37 = fptr[37] > 0;
        u.t.b38 = fptr[38] > 0;
        u.t.b39 = fptr[39] > 0;
        u.t.b40 = fptr[40] > 0;
        u.t.b41 = fptr[41] > 0;
        u.t.b42 = fptr[42] > 0;
        u.t.b43 = fptr[43] > 0;
        u.t.b44 = fptr[44] > 0;
        u.t.b45 = fptr[45] > 0;
        u.t.b46 = fptr[46] > 0;
        u.t.b47 = fptr[47] > 0;
        u.t.b48 = fptr[48] > 0;
        u.t.b49 = fptr[49] > 0;
        u.t.b50 = fptr[50] > 0;
        u.t.b51 = fptr[51] > 0;
        u.t.b52 = fptr[52] > 0;
        u.t.b53 = fptr[53] > 0;
        u.t.b54 = fptr[54] > 0;
        u.t.b55 = fptr[55] > 0;
        u.t.b56 = fptr[56] > 0;
        u.t.b57 = fptr[57] > 0;
        u.t.b58 = fptr[58] > 0;
        u.t.b59 = fptr[59] > 0;
        u.t.b60 = fptr[60] > 0;
        u.t.b61 = fptr[61] > 0;
        u.t.b62 = fptr[62] > 0;
        u.t.b63 = fptr[63] > 0;

        *buf = u.u64;
    }
};

inline void binarize(const bnn::Mat &float_mat, bnn::Mat &binary_mat) {
    /**
     * This is the bit-packing for tensor of less than 128 channels.
     */
    BNN_ASSERT(
        float_mat.w * float_mat.c > 0 && float_mat.w * float_mat.c % 64 == 0,
        float_mat.w * float_mat.c);
    BNN_ASSERT(float_mat.c / 64 == binary_mat.c && float_mat.c % 64 == 0,
               "float_mat.c ", float_mat.c, ", binary_mat.c ", binary_mat.c);
       
    int size = float_mat.n * float_mat.h * (float_mat.w * float_mat.c / 64);
    pack_64_bitfield_functor func((float*)float_mat.data, (uint64_t*)binary_mat.data);
    thrust::counting_iterator<int> iter(0);
    thrust::for_each(iter, iter + size, func);
    hipDeviceSynchronize();
}

void Binarize::forward_impl() const { binarize(*input_mat, *output_mat); }

}  // namespace bnn
