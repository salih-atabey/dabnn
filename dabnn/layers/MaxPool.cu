#include "hip/hip_runtime.h"
// Copyright 2019 JD.com Inc. JD AI

#include "MaxPool.h"

#include <limits>

#include <dabnn/net.h>
#include <dabnn/pad.h>

namespace bnn {

template<typename T>
struct maxpool_functor {
    const T* input;
    const size_t pad_h, pad_w, stride_h, stride_w, kernel_h, kernel_w, output_w, input_hstep;
    const int input_h, input_w, input_c;

    maxpool_functor(const T* _input, const size_t _pad_h, const size_t _pad_w,
            const size_t _stride_h, const size_t _stride_w, const size_t _kernel_h,
            const size_t _kernel_w, const int _output_w, const int _input_h,
            const int _input_w, const int _input_c, const size_t _input_hstep)
        : input(_input), pad_h(_pad_h), pad_w(_pad_w), stride_h(_stride_h),
            stride_w(_stride_w), kernel_h(_kernel_h), kernel_w(_kernel_w),
            output_w(_output_w), input_h(_input_h), input_w(_input_w), input_c(_input_c), input_hstep(_input_hstep) {}

    __host__ __device__
    T operator()(const int output_idx) const {
        const int output_y = output_idx / output_w;
        const int output_x = output_idx % output_w;

        int input_y = output_y * stride_h - pad_h;
        int input_x = output_x * stride_w - pad_w;

        const T* init_ptr = input + input_y * input_hstep;

        T m = init_ptr[input_x * input_c];
        thrust::maximum<T> max;

        for (int kh = 0; kh < kernel_h; kh++) {
            int y = input_y + kh;
            if (y >= 0 && y < input_h) {
                // const T* input_ptr = input.point<T>(y, 0);
                const T* input_ptr = input + y * input_hstep;
                for (int kw = 0; kw < kernel_w; kw++) {
                    int x = input_x + kw;
                    if (!(y < 0 || y >= input_h || x < 0 || x >= input_w)) {
                        const auto val = input_ptr[x * input_c];
                        m = max(m, val);
                    }
                }
            }
        }

        return m;
    }
};

#ifdef __ARM_NEON
void maxpool2x2(const bnn::Mat &input, bnn::Mat &output, const int stride_h = 1,
                const int stride_w = 1) {
    FORZ(h, output.h) {
        FORZ(w, output.w) {
            const float *ptr0 =
                input.point<float>(h * stride_h + 0, w * stride_w + 0);
            const float *ptr1 =
                input.point<float>(h * stride_h + 0, w * stride_w + 1);
            const float *ptr2 =
                input.point<float>(h * stride_h + 1, w * stride_w + 0);
            const float *ptr3 =
                input.point<float>(h * stride_h + 1, w * stride_w + 1);
            float *output_ptr = output.point<float>(h, w);
            size_t nn = input.c >> 2;
#ifdef __aarch64__
            asm volatile(
                "0:     \n"
                "ld1    {v0.4s}, [%0], #16      \n"
                "prfm   pldl1keep, [%0, #128]   \n"
                "ld1    {v1.4s}, [%1], #16      \n"
                "prfm   pldl1keep, [%1, #128]   \n"
                "ld1    {v2.4s}, [%2], #16      \n"
                "prfm   pldl1keep, [%2, #128]   \n"
                "ld1    {v3.4s}, [%3], #16      \n"
                "prfm   pldl1keep, [%3, #128]   \n"
                "fmax   v0.4s, v0.4s, v1.4s     \n"
                "fmax   v2.4s, v2.4s, v3.4s     \n"
                "fmax   v0.4s, v0.4s, v2.4s     \n"
                "subs   %5, %5, #1              \n"
                "st1    {v0.4s}, [%4], #16      \n"
                "bne    0b                      \n"

                : "+r"(ptr0),        // %0
                  "+r"(ptr1),        // %1
                  "+r"(ptr2),        // %2
                  "+r"(ptr3),        // %3
                  "+r"(output_ptr),  // %4
                  "+r"(nn)           // %5
                :
                : "cc", "memory", "v0", "v1", "v2", "v3", "v4", "v5", "v6",
                  "v7", "v8", "v9", "v10", "v11", "v12");
#else   // __aarch64__
            asm volatile(
                "0:     \n"
                "vld1.32    q0, [%0]!       \n"
                "pld    [%0, #128]          \n"
                "vld1.32    q1, [%1]!       \n"
                "pld    [%1, #128]          \n"
                "vld1.32    q2, [%2]!       \n"
                "pld    [%2, #128]          \n"
                "vld1.32    q3, [%3]!       \n"
                "pld    [%3, #128]          \n"
                "vmax.f32   q0, q0, q1      \n"
                "vmax.f32   q2, q2, q3      \n"
                "vmax.f32   q0, q0, q2      \n"
                "subs   %5, %5, #1          \n"
                "vst1.32    q0, [%4]!       \n"
                "bne    0b                  \n"

                : "+r"(ptr0),        // %0
                  "+r"(ptr1),        // %1
                  "+r"(ptr2),        // %2
                  "+r"(ptr3),        // %3
                  "+r"(output_ptr),  // %4
                  "+r"(nn)           // %5
                :
                : "cc", "memory", "q0", "q1", "q2", "q3");
#endif  // __aarch64__
        }
    }
}

void maxpool3x3(const bnn::Mat &input, bnn::Mat &output, const int stride_h = 1,
                const int stride_w = 1) {
    FORZ(h, output.h) {
        FORZ(w, output.w) {
            const float *ptr0 =
                input.point<float>(h * stride_h + 0, w * stride_w + 0);
            const float *ptr1 =
                input.point<float>(h * stride_h + 0, w * stride_w + 1);
            const float *ptr2 =
                input.point<float>(h * stride_h + 0, w * stride_w + 2);
            const float *ptr3 =
                input.point<float>(h * stride_h + 1, w * stride_w + 0);
            const float *ptr4 =
                input.point<float>(h * stride_h + 1, w * stride_w + 1);
            const float *ptr5 =
                input.point<float>(h * stride_h + 1, w * stride_w + 2);
            const float *ptr6 =
                input.point<float>(h * stride_h + 2, w * stride_w + 0);
            const float *ptr7 =
                input.point<float>(h * stride_h + 2, w * stride_w + 1);
            const float *ptr8 =
                input.point<float>(h * stride_h + 2, w * stride_w + 2);
            float *output_ptr = output.point<float>(h, w);
            size_t nn = input.c >> 2;
#ifdef __aarch64__
            asm volatile(
                "0:     \n"
                "ld1    {v0.4s}, [%0], #16      \n"
                "prfm   pldl1keep, [%0, #128]   \n"
                "ld1    {v1.4s}, [%1], #16      \n"
                "prfm   pldl1keep, [%1, #128]   \n"
                "ld1    {v2.4s}, [%2], #16      \n"
                "prfm   pldl1keep, [%2, #128]   \n"
                "ld1    {v3.4s}, [%3], #16      \n"
                "prfm   pldl1keep, [%3, #128]   \n"
                "fmax   v0.4s, v0.4s, v1.4s     \n"
                "ld1    {v4.4s}, [%4], #16      \n"
                "prfm   pldl1keep, [%4, #128]   \n"
                "fmax   v2.4s, v2.4s, v3.4s     \n"
                "ld1    {v5.4s}, [%5], #16      \n"
                "prfm   pldl1keep, [%5, #128]   \n"
                "ld1    {v6.4s}, [%6], #16      \n"
                "prfm   pldl1keep, [%6, #128]   \n"
                "fmax   v4.4s, v4.4s, v5.4s     \n"
                "ld1    {v7.4s}, [%7], #16      \n"
                "prfm   pldl1keep, [%7, #128]   \n"
                "ld1    {v8.4s}, [%8], #16      \n"
                "prfm   pldl1keep, [%8, #128]   \n"
                "fmax   v2.4s, v2.4s, v8.4s     \n"
                "fmax   v6.4s, v6.4s, v7.4s     \n"
                "fmax   v0.4s, v0.4s, v2.4s     \n"
                "fmax   v4.4s, v4.4s, v6.4s     \n"
                "subs   %10, %10, #1              \n"
                "fmax   v0.4s, v0.4s, v4.4s     \n"
                "st1    {v0.4s}, [%9], #16      \n"
                "bne    0b                      \n"

                : "+r"(ptr0),        // %0
                  "+r"(ptr1),        // %1
                  "+r"(ptr2),        // %2
                  "+r"(ptr3),        // %3
                  "+r"(ptr4),        // %4
                  "+r"(ptr5),        // %5
                  "+r"(ptr6),        // %6
                  "+r"(ptr7),        // %7
                  "+r"(ptr8),        // %8
                  "+r"(output_ptr),  // %9
                  "+r"(nn)           // %10
                :
                : "cc", "memory", "v0", "v1", "v2", "v3", "v4", "v5", "v6",
                  "v7", "v8", "v9", "v10", "v11", "v12");
#else
            asm volatile(
                "0:     \n"
                "vld1.32    q0, [%0]!       \n"
                "pld    [%0, #128]          \n"
                "vld1.32    q1, [%1]!       \n"
                "pld    [%1, #128]          \n"
                "vld1.32    q2, [%2]!       \n"
                "pld    [%2, #128]          \n"
                "vld1.32    q3, [%3]!       \n"
                "pld    [%3, #128]          \n"
                "vmax.f32   q0, q0, q1      \n"
                "vld1.32    q4, [%4]!       \n"
                "pld    [%4, #128]          \n"
                "vmax.f32   q2, q2, q3      \n"
                "vld1.32    q5, [%5]!       \n"
                "pld    [%5, #128]          \n"
                "vld1.32    q6, [%6]!       \n"
                "pld    [%6, #128]          \n"
                "vmax.f32   q4, q4, q5      \n"
                "vld1.32    q7, [%7]!       \n"
                "pld    [%7, #128]          \n"
                "vld1.32    q8, [%8]!       \n"
                "pld    [%8, #128]          \n"
                "vmax.f32   q2, q2, q8      \n"
                "vmax.f32   q6, q6, q7      \n"
                "vmax.f32   q0, q0, q2      \n"
                "subs       %10, %10, #1    \n"
                "vmax.f32   q4, q4, q6      \n"
                "vmax.f32   q0, q0, q4      \n"
                "vst1.32    q0, [%9]!       \n"
                "bne    0b                  \n"

                : "+r"(ptr0),        // %0
                  "+r"(ptr1),        // %1
                  "+r"(ptr2),        // %2
                  "+r"(ptr3),        // %3
                  "+r"(ptr4),        // %4
                  "+r"(ptr5),        // %5
                  "+r"(ptr6),        // %6
                  "+r"(ptr7),        // %7
                  "+r"(ptr8),        // %8
                  "+r"(output_ptr),  // %9
                  "+r"(nn)           // %10
                :
                : "cc", "memory", "q0", "q1", "q2", "q3", "q4", "q5", "q6",
                  "q7", "q8");
#endif
        }
    }
}
#endif  // __ARM_NEON

void max_pool_fallback(const bnn::Mat &input, const size_t pad_h,
                       const size_t pad_w, const size_t stride_h,
                       const size_t stride_w, const size_t kernel_h,
                       const size_t kernel_w, bnn::Mat &output) {
    const int output_h =
        (input.h + 2 * pad_h - ((kernel_h - 1) + 1)) / stride_h + 1;
    const int output_w =
        (input.w + 2 * pad_w - ((kernel_w - 1) + 1)) / stride_w + 1;

    BNN_ASSERT(input.w * input.c * input.elemsize % 16 == 0, "Not align");
    BNN_ASSERT(output.w * output.c * output.elemsize % 16 == 0, "Not align");
    BNN_ASSERT(input.data_type == input.data_type, "Mismatch datatype");

    const int input_h = input.h;
    const int input_w = input.w;
    const int input_c = input.c;
    const size_t input_hstep = input.hstep;

    thrust::counting_iterator<int> idx_begin(0);
    thrust::counting_iterator<int> idx_end = idx_begin + output_h * output_w;
    
    if (input.data_type == DataType::Float) { 
        maxpool_functor<float> func(input, pad_h, pad_w, stride_h, stride_w, kernel_h, kernel_w, output_w, input_h, input_w, input_c, input_hstep);
        thrust::device_vector<float> output_values(output_h * output_w * input_c);
        thrust::transform(thrust::device, idx_begin, idx_end, output.begin<float>(), func);
    } else if (input.data_type == DataType::Bit) {
        maxpool_functor<uint64_t> func(input, pad_h, pad_w, stride_h, stride_w, kernel_h, kernel_w, output_w, input_h, input_w, input_c, input_hstep);
        thrust::device_vector<uint64_t> output_values(output_h * output_w * input_c);
        thrust::transform(thrust::device, idx_begin, idx_end, output.begin<uint64_t>(), func);
    } else {
        throw std::invalid_argument("Unknown datatype");
    }
}

MaxPool::MaxPool(NetCP net, const std::string &name, css input, css output,
                 int kernel_h, int kernel_w, int pad_h, int pad_w, int stride_h,
                 int stride_w)
    : Layer(net, name, "MaxPool"),
      input_mat(mat(input)),
      output_mat(mat(output)),
      kernel_h(kernel_h),
      kernel_w(kernel_w),
      pad_h(pad_h),
      pad_w(pad_w),
      stride_h(stride_h),
      stride_w(stride_w) {
    auto &mat_map = net.lock()->mat_map_;
    const auto &pad_name = "pad_for_" + output + "_cal";
    if (mat_map.find(pad_name) == mat_map.end()) {
        auto &input_mat = *mat_map[input];
        mat_map[pad_name] = std::make_shared<Mat>(
            input_mat.h + pad_h * 2, input_mat.w + pad_w * 2, input_mat.c,
            input_mat.data_type, pad_name);
    }
    padded_mat = mat_map[pad_name];
}
void MaxPool::forward_impl() const {
#ifdef __ARM_NEON
    if (kernel_h == 3 && kernel_w == 3) {
        // std::numeric_limits<float>::min() is the closest value to 0, so we
        // uses -max()
        pad(*input_mat, pad_h, pad_w, *padded_mat,
            -std::numeric_limits<float>::max());
        maxpool3x3(*padded_mat, *output_mat, stride_h, stride_w);
    } else if (kernel_h == 2 && kernel_w == 2) {
        pad(*input_mat, pad_h, pad_w, *padded_mat,
            -std::numeric_limits<float>::max());
        maxpool2x2(*padded_mat, *output_mat, stride_h, stride_w);
    } else {
        max_pool_fallback(*input_mat, pad_h, pad_w, stride_h, stride_w,
                          kernel_h, kernel_w, *output_mat);
    }
#else
    max_pool_fallback(*input_mat, pad_h, pad_w, stride_h, stride_w, kernel_h,
                      kernel_w, *output_mat);
#endif  // __aarch64__
}

std::string MaxPool::to_str() const {
    std::stringstream ss;
    ss << type_ << ", ";
    PNT_TO(ss, kernel_h, kernel_w, stride_h, stride_w);
    return ss.str();
}

}  // namespace bnn
